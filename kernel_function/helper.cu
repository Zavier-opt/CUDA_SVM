
#include <hip/hip_runtime.h>
#include<float.h>
#include <vector>

using namespace std;

void findSupportVector(vector<float> &h_e, vector<int> &h_Ilow,vector<int> &h_Iup,int *low, int *up){
    double min = DBL_MAX;
    double max = -DBL_MAX;
    for(int i=0;i<h_e.size();i++){
        if(h_Ilow[i]==1){ // it is in low group
            if(h_e[i]>max){
                max = h_e[i];
                *low = i;
            }
        }
        if(h_Iup[i]==1){ // it is in up group
            if(h_e[i]<min){
                min = h_e[i];
                *up = i;
            }
        }
    }
}