
#include <hip/hip_runtime.h>
#include<float.h>
#include <vector>


using namespace std;

void findSupportVector(vector<float> &h_e, vector<int> &h_Ilow,vector<int> &h_Iup,int *low, int *up){
    double min = DBL_MAX;
    double max = -DBL_MAX;
    for(int i=0;i<h_e.size();i++){
        if(h_Ilow[i]==1){ // it is in low group
            if(h_e[i]>max){
                max = h_e[i];
                *low = i;
            }
        }
        if(h_Iup[i]==1){ // it is in up group
            if(h_e[i]<min){
                min = h_e[i];
                *up = i;
            }
        }
    }
}

void make_prediction(vector<int> &predict_res, int low, int up,int numOfData,float C,vector<float> &kernel_up, vector<float> &kernel_low, vector<float> &alpha, vector<int> &y) {
    // cal b
    float b = 0;
    for (int i = 0; i < numOfData; i++) {
        if (alpha[i] < 0 || alpha[i] >= C) {
            continue;
        }
        float temp = y[i] - alpha[low] * y[low] * kernel_low[i] + alpha[up] * y[up] * kernel_up[i];
        b = b + temp;
    }
    for (int i = 0; i < numOfData; i++) {
        float decision = alpha[low] * y[low] * kernel_low[i] + alpha[up] * y[up] * kernel_up[i] + b;
        if (decision >= 0) {
            predict_res[i] = 1;
        }
        else {
            predict_res[i] = -1;
        }
    }
}

float accuracy(vector<int>& predict_res, vector<int>& y, int numOfData) {
    float countCorrect = 0;
    for (int i = 0; i < numOfData; i++) {
        if (predict_res[i] == y[i]) {
            countCorrect++;
        }
    }
    return countCorrect / numOfData;
}