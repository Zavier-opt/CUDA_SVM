
#include <hip/hip_runtime.h>
#include "math.h"
#include <stdio.h>

__global__ void smo_kernel_initial(float *d_x, int *d_y, float *d_e, float *d_alpha, int *d_Iup, int *d_Ilow, int numOfData, int numOfAttr,float C);
__global__ void calculate_kernel_update_alpha(int low, int up, float *kernel_value, float *d_x, int *d_y, float *d_e,float *d_alpha, int *d_Iup, int *d_Ilow, int numOfData, int numOfAttr,bool cal_low, bool cal_up, int row_low, int row_up, char kernel_function_name[4], float Gamma, float C);
__device__ void cal_put_kernel(float x[], float support_vector[], int numOfAttr,int numOfData, int index, float *kernel_value, int row_pos, char kernel_function_name[4], float Gamma);
__device__ bool check(char name[4], char kernel_function_name[4]);
__device__ float RBF(float x[], float support_vector[],int numOfAttr,float Gamma);
__device__ float LINEAR(float x[], float support_vector[], int numOfAttr);
__device__ void update_alpha_e(int low, int up, float *d_e,float *d_alpha, int *d_y, float *kernel_value, int row_low, int row_up, int index, int numOfData);
__device__ int divideGroup(int y, float alpha, float C, bool isUp);
__global__ void write_kernel_to_memory(float *d_kernel_up, float *d_kernel_low, int row_up, int row_low, float *kernel_value,int numOfData);



__global__ void smo_kernel_initial(float *d_x, int *d_y, float *d_e, float *d_alpha, int *d_Iup, int *d_Ilow, int numOfData, int numOfAttr,float C ){
    // value preparation
    float C_local = C;
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    // put data from global memory to shared memory
    if(index<numOfData){
        d_e[index] = -d_y[index];
        d_alpha[index] = 0;
        d_Iup[index] = divideGroup(d_y[index], d_alpha[index], C_local,true); // true: detect up; false: detect low
        d_Ilow[index] = divideGroup(d_y[index], d_alpha[index], C_local,false);// the value is 1 or 0;
        //printf("%d ",index);
   }
}
__global__ void calculate_kernel_update_alpha(int low, int up, float *kernel_value, float *d_x, int *d_y, float *d_e,float *d_alpha, int *d_Iup, int *d_Ilow, int numOfData, int numOfAttr,bool cal_low, bool cal_up, int row_low, int row_up, char kernel_function_name[4], float Gamma, float C){
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    if(index<numOfData){
        //printf("%d", index);
        float local_x_data[256];
        float local_low_data[256];
        float local_up_data[256];
        float C_local = C;
        for(int i=0;i<numOfAttr;i++){
            local_x_data[i] = d_x[index*numOfAttr+i];
            local_low_data[i] = d_x[low*numOfAttr+i];
            local_up_data[i] = d_x[up*numOfAttr+i];
        }

 //       if (index == 3) {
 //          for (int i = 0; i < numOfAttr; i++) {
 //               printf("%.2f ", local_x_data[i]);
 //           }
 //           printf("\n");
 //           for (int i = 0; i < numOfAttr; i++) {
 //              printf("%.2f ", local_low_data[i]);
 //           }
 //           printf("\n");
 //           for (int i = 0; i < numOfAttr; i++) {
 //               printf("%.2f ", local_up_data[i]);
 //           }
 //           printf("\n");
 //       }

        if(cal_low){
            //printf("%d", index);
            cal_put_kernel(local_x_data,local_low_data, numOfAttr,numOfData,index,kernel_value,row_low,kernel_function_name, Gamma);
        }
        if(cal_up){
            cal_put_kernel(local_x_data,local_up_data, numOfAttr,numOfData,index,kernel_value,row_up,kernel_function_name, Gamma);
        }

        update_alpha_e(low,up,d_e,d_alpha,d_y, kernel_value,row_low,row_up,index, numOfData);

        d_Iup[index] = divideGroup(d_y[index], d_alpha[index], C_local,true); // true: detect up; false: detect low
        d_Ilow[index] = divideGroup(d_y[index], d_alpha[index], C_local,false);// the value is 1 or 0;
    }
}
__device__ void cal_put_kernel(float x[], float support_vector[], int numOfAttr,int numOfData, int index, float *kernel_value, int row_pos, char kernel_function_name[4], float Gamma){
    //printf("%d", index);
    char rbf[4] = "RBF";
    char lin[4] = "LIN";
    float value;
    //if(check(rbf,kernel_function_name)){
    //    value = RBF(x,support_vector,numOfAttr,Gamma);
    //}else{
    //    value = LINEAR(x,support_vector,numOfAttr);
    //}
    //bool temp = check(rbf, kernel_function_name);
    //printf("temp:%d\n", temp);

    value = RBF(x, support_vector, numOfAttr, Gamma);
    kernel_value[row_pos*numOfData+index]=value;
    
    //printf("pos:%d\n", row_pos * numOfData + index);
    //printf("value:%.4f\n", value);

}
__device__ bool check(char name[4], char kernel_function_name[4]){
    bool res = true;
    for(int i=0;i<4;i++){
        if(name[i]!=kernel_function_name[i]){
            res = false;
        }
    }
    return res;
}
__device__ float RBF(float x[], float support_vector[],int numOfAttr,float Gamma){
    //int index2 = blockIdx.x * blockDim.x + threadIdx.x;
    //if (index2 == 3) {
    //    for (int i = 0; i < numOfAttr; i++) {
    //        printf("%.2f ", x[i]);
    //    }
    //    for (int i = 0; i < numOfAttr; i++) {
    //        printf("%.2f ", support_vector[i]);
    //    }
    //}
    //printf("\n");
    float sum = 0;
    for(int i=0;i<numOfAttr;i++){
        sum+=(x[i]-support_vector[i])* (x[i] - support_vector[i]);
        //printf("%.2f\n", sum);
    }
    
    return exp(-sum*Gamma);
}

__device__ float LINEAR(float x[], float support_vector[], int numOfAttr){
    float sum=0;
    for(int i=0;i<numOfAttr;i++){
        sum+=x[i]*support_vector[i];
    }
    return sum;
}
 
__device__ void update_alpha_e(int low, int up, float *d_e,float *d_alpha, int *d_y, float *kernel_value, int row_low, int row_up, int index, int numOfData){
    int index3 = blockIdx.x * blockDim.x + threadIdx.x;
    int y_up = d_y[up];
    int y_low = d_y[low];
    float alpha_low = d_alpha[low];
    float alpha_up = d_alpha[up];
    float alpha_low_new;
    float alpha_up_new;
    int s = y_up * y_low;
    float k_low_low = kernel_value[row_low * numOfData + low];
    float k_up_up = kernel_value[row_up * numOfData + up];
    float k_low_up = kernel_value[row_low * numOfData + up];
    float miu = k_low_low + k_up_up - 2 * k_low_up;
    if (index3 == 0) {
        printf("y_up:%d", y_up);
        printf("y_low:%d", y_low);
        printf("e_low:%0.2f", d_e[low]);
        printf("e_up:%0.2f", d_e[up]);
        printf("miu: %.2f", miu);
    }
    alpha_up_new = alpha_up + y_up * (d_e[low] - d_e[up]) / miu;
    alpha_low_new = alpha_low + s * (alpha_up - alpha_up_new);
    float kernel_low_i = kernel_value[row_low * numOfData + index];
    float kernel_up_i = kernel_value[row_up * numOfData + index];
    float e_i = d_e[index];
    d_e[index] = e_i + (alpha_low_new - alpha_low) * y_low * kernel_low_i + (alpha_up_new - alpha_up) * y_up * kernel_up_i;
    __syncthreads();
    if(index == row_up){
        d_alpha[up]=alpha_up_new;
        d_alpha[low]=alpha_low_new;
    }
    __syncthreads();
}

__device__ int divideGroup(int y, float alpha, float C, bool isUp){
   if(alpha>0 && alpha<C){
       return 1;
   }
   int res;
   if(isUp){
       if((y==1&&alpha==0)||(y==-1&&alpha==C)){
           res = 1;
       }else{
           res = 0;
       }
       return res;
   }else{
       if((y==1&&alpha==C)||(y==-1&&alpha==0)){
           res = 1;
       }else{
           res = 0;
       }
       return res;
   }
  
 
}

__global__ void write_kernel_to_memory(float *d_kernel_up, float *d_kernel_low, int row_up, int row_low, float *kernel_value,int numOfData){
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    if(index<numOfData){
        d_kernel_up[index]=kernel_value[row_up*numOfData+index];
        d_kernel_low[index]=kernel_value[row_low*numOfData+index];
    }
}