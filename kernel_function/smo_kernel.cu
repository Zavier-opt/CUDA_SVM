#include "hip/hip_runtime.h"
#include "string.h"
#include "math.h"
__global__ void smo_kernel_initial(float *d_x, int *d_y, float *d_e, float *d_alpha, int *d_Iup, int *d_Ilow, int numOfData, int numOfAttr,float C ){
    // value preparation
    float C_local = C;
    int index = blockIdx.x*blockDim.x+threadIdx.x;

    // put data from global memory to shared memory
    if(index<numOfData){
        d_e[index] = -d_y[index];
        d_alpha[index] = 0;
        d_Iup[index] = divideGroup(d_y[index], alpha[index], C_local,true); // true: detect up; false: detect low
        d_Ilow[index] = divideGroup(d_y[index], alpha[index], C_local,false);// if it is up/low, the value is 1, otherwise 0;
       
   }
}
__global__ void calculate_kernel_update_alpha(int low, int up, float *kernel_value, float *d_x, float *d_e,float *d_alpha, int numOfData, int numOfAttr,bool cal_low, bool cal_up, int row_low, int row_up, char[] kernel_function, float gamma){
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    if(index<numOfData){
        float local_x_data[numOfAttr];
        float local_low_data[numOfAttr];
        float local_up_data[numOfAttr];
        for(int i=0;i<local_data;i++){
            local_x_data[i] = d_x[index*numOfAttr+i];
            local_low_data[i] = d_x[low*numOfAttr+i];
            local_up_data[i] = d_x[up*numOfAttr+i];
        }
        if(cal_low){
            cal_put_kernel(local_x_data,local_low_data, numOfAttr,index,kernel_value,row_low,kernel_function, gamma);
        }
        if(cal_up){
            cal_put_kernel(local_x_data,local_up_data, numOfAttr,index,kernel_value,row_up,kernel_function, gamma);
        }

        update_alpha_e(d_e,d_alpha,kernel_value,row_low,row_up,index);
    }
}
__device__ void cal_put_kernel(float x[], float support_vector[], int numOfAttr, int index, float *kernel_value, int row_pos, char[] kernel_function, float gamma){
    char[] rbf = "RBF";
    char[] lin = "LINEAR";
    float value;
    if(strcmp(rbf,kernel_function)==0){
        value = RBF(x,support_vector,numOfAttr,gamma);
    }else{
        value = LINEAR(x,support_vector,numOfAttr);
    }
    kernel_value[row_pos*numOfData+index]=value;

}

__device__ float RBF(float x[], float support_vector[],int numOfAttr,float gamma){
    float sum = 0;
    for(int i=0;i<numOfAttr;i++){
        sum+=pow((x[i]-support_vector[i]),2);
    }
    return exp(-sum*gamma);
}

__device__ float LINEAR(float x[], float support_vector[], int numOfAttr){
    float sum=0;
    for(int i=0;i<numOfAttr;i++){
        sum+=x[i]*support_vector[i];
    }
    return sum;
}
 
__device__ void update_alpha_e(float *d_e,float *d_alpha, float *kernel_value, int row_low, int row_up, int index){
    float low_kernel_value = 
    float up_kernel_value =
}

__device__ int divideGroup(int y, float alpha, float C, bool isUp){
   if(alpha>0 && alpha<C){
       return 1;
   }
   int res;
   if(isUp){
       if((y==1&&alpha==0)||(y==-1&&alpha==C)){
           res = 1;
       }else{
           res = 0;
       }
       return res;
   }else{
       if((y==1&&alpha==C)||(y==-1&&alpha==0)){
           res = 1;
       }else{
           res = 0;
       }
       return res;
   }
  
 
}
