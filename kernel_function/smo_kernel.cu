
#include <hip/hip_runtime.h>
#include "math.h"
__global__ void smo_kernel_initial(float *d_x, int *d_y, float *d_e, float *d_alpha, int *d_Iup, int *d_Ilow, int numOfData, int numOfAttr,float C);
__global__ void calculate_kernel_update_alpha(int low, int up, float *kernel_value, float *d_x, int *d_y, float *d_e,float *d_alpha, int *d_Iup, int *d_Ilow, int numOfData, int numOfAttr,bool cal_low, bool cal_up, int row_low, int row_up, char kernel_function_name[4], float Gamma, float C);
__device__ void cal_put_kernel(float x[], float support_vector[], int numOfAttr,int numOfData, int index, float *kernel_value, int row_pos, char kernel_function_name[4], float Gamma);
__device__ bool check(char name[4], char kernel_function_name[4]);
__device__ float RBF(float x[], float support_vector[],int numOfAttr,float Gamma);
__device__ float LINEAR(float x[], float support_vector[], int numOfAttr);
__device__ void update_alpha_e(float *d_e,float *d_alpha, int *d_y, float *kernel_value, int row_low, int row_up, int index, int numOfData);
__device__ int divideGroup(int y, float alpha, float C, bool isUp);
__global__ void write_kernel_to_memory(float *d_kernel_up, float *d_kernel_low, int row_up, int row_low, float *kernel_value,int numOfData);



__global__ void smo_kernel_initial(float *d_x, int *d_y, float *d_e, float *d_alpha, int *d_Iup, int *d_Ilow, int numOfData, int numOfAttr,float C ){
    // value preparation
    float C_local = C;
    int index = blockIdx.x*blockDim.x+threadIdx.x;

    // put data from global memory to shared memory
    if(index<numOfData){
        d_e[index] = -d_y[index];
        d_alpha[index] = 0;
        d_Iup[index] = divideGroup(d_y[index], d_alpha[index], C_local,true); // true: detect up; false: detect low
        d_Ilow[index] = divideGroup(d_y[index], d_alpha[index], C_local,false);// the value is 1 or 0;
       
   }
}
__global__ void calculate_kernel_update_alpha(int low, int up, float *kernel_value, float *d_x, int *d_y, float *d_e,float *d_alpha, int *d_Iup, int *d_Ilow, int numOfData, int numOfAttr,bool cal_low, bool cal_up, int row_low, int row_up, char kernel_function_name[4], float Gamma, float C){
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    if(index<numOfData){
        float local_x_data[256];
        float local_low_data[256];
        float local_up_data[256];
        float C_local = C;
        for(int i=0;i<numOfAttr;i++){
            local_x_data[i] = d_x[index*numOfAttr+i];
            local_low_data[i] = d_x[low*numOfAttr+i];
            local_up_data[i] = d_x[up*numOfAttr+i];
        }
        if(cal_low){
            cal_put_kernel(local_x_data,local_low_data, numOfAttr,numOfData,index,kernel_value,row_low,kernel_function_name, Gamma);
        }
        if(cal_up){
            cal_put_kernel(local_x_data,local_up_data, numOfAttr,numOfData,index,kernel_value,row_up,kernel_function_name, Gamma);
        }
        update_alpha_e(d_e,d_alpha,d_y, kernel_value,row_low,row_up,index, numOfData);

        d_Iup[index] = divideGroup(d_y[index], d_alpha[index], C_local,true); // true: detect up; false: detect low
        d_Ilow[index] = divideGroup(d_y[index], d_alpha[index], C_local,false);// the value is 1 or 0;
    }
}
__device__ void cal_put_kernel(float x[], float support_vector[], int numOfAttr,int numOfData, int index, float *kernel_value, int row_pos, char kernel_function_name[4], float Gamma){
    char rbf[4] = "RBF";
    char lin[4] = "LIN";
    float value;
    if(check(rbf,kernel_function_name)){
        value = RBF(x,support_vector,numOfAttr,Gamma);
    }else{
        value = LINEAR(x,support_vector,numOfAttr);
    }
    kernel_value[row_pos*numOfData+index]=value;

}
__device__ bool check(char name[4], char kernel_function_name[4]){
    bool res = true;
    for(int i=0;i<4;i++){
        if(name[i]!=kernel_function_name[i]){
            res = false;
        }
    }
    return res;
}
__device__ float RBF(float x[], float support_vector[],int numOfAttr,float Gamma){
    float sum = 0;
    for(int i=0;i<numOfAttr;i++){
        sum+=pow((x[i]-support_vector[i]),2);
    }
    return exp(-sum*Gamma);
}

__device__ float LINEAR(float x[], float support_vector[], int numOfAttr){
    float sum=0;
    for(int i=0;i<numOfAttr;i++){
        sum+=x[i]*support_vector[i];
    }
    return sum;
}
 
__device__ void update_alpha_e(float *d_e,float *d_alpha, int *d_y, float *kernel_value, int row_low, int row_up, int index, int numOfData){
    int y_up = d_y[row_up];
    int y_low = d_y[row_low];
    float alpha_low = d_alpha[row_low];
    float alpha_up = d_alpha[row_up];
    if(index == row_up){
        int s = y_up*y_low;
        float k_low_low = kernel_value[row_low*numOfData+row_low];
        float k_up_up = kernel_value[row_up*numOfData+row_up];
        float k_low_up = kernel_value[row_low*numOfData+row_up];
        float miu = k_low_low+k_up_up-2*k_low_up;

        float alpha_up_new = d_alpha[row_up]+y_up*(d_e[row_low]-d_e[row_up])/miu;
        float alpha_low_new = d_alpha[row_low]+s*(d_alpha[row_up]-alpha_up_new);

        d_alpha[row_up]=alpha_up_new;
        d_alpha[row_low]=alpha_low_new;
    }
    float alpha_low_new = d_alpha[row_low];
    float alpha_up_new = d_alpha[row_up];
    float kernel_low_i = kernel_value[row_low*numOfData+index];
    float kernel_up_i = kernel_value[row_up*numOfData+index];
    float e_i = d_e[index];
    d_e[index] = e_i + (alpha_low_new-alpha_low)*y_low*kernel_low_i + (alpha_up_new-alpha_up)*y_up*kernel_up_i;
    
}

__device__ int divideGroup(int y, float alpha, float C, bool isUp){
   if(alpha>0 && alpha<C){
       return 1;
   }
   int res;
   if(isUp){
       if((y==1&&alpha==0)||(y==-1&&alpha==C)){
           res = 1;
       }else{
           res = 0;
       }
       return res;
   }else{
       if((y==1&&alpha==C)||(y==-1&&alpha==0)){
           res = 1;
       }else{
           res = 0;
       }
       return res;
   }
  
 
}

__global__ void write_kernel_to_memory(float *d_kernel_up, float *d_kernel_low, int row_up, int row_low, float *kernel_value,int numOfData){
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    if(index<numOfData){
        d_kernel_up[index]=kernel_value[row_up*numOfData+index];
        d_kernel_low[index]=kernel_value[row_low*numOfData+index];
    }
}