#include "hip/hip_runtime.h"
__global__ void smo_kernel_initial(float *d_x, int *d_y, float *d_e, float *d_alpha, int *d_Iup, int *d_Ilow, int numOfData, int numOfAttr,float C ){
   if(initial){
       // value preparation
       float C_local = C;
       int index = blockIdx.x*blockDim.x+threadIdx.x;
 
       // put data from global memory to shared memory
       if(index<numOfData){
           d_e[index] = -d_y[index];
           d_alpha[index] = 0;
           d_Iup[index] = divideGroup(d_y[index], alpha[index], C_local,true); // true: detect up; false: detect low
           d_Ilow[index] = divideGroup(d_y[index], alpha[index], C_local,false);// if it is up/low, the value is 1, otherwise 0;
       }
 
   }
 
}
 
__device__ int divideGroup(int y, float alpha, float C, bool isUp){
   if(alpha>0 && alpha<C){
       return 1;
   }
   int res;
   if(isUp){
       if((y==1&&alpha==0)||(y==-1&&alpha==C)){
           res = 1;
       }else{
           res = 0;
       }
       return res;
   }else{
       if((y==1&&alpha==C)||(y==-1&&alpha==0)){
           res = 1;
       }else{
           res = 0;
       }
       return res;
   }
  
 
}
