#include "hip/hip_runtime.h"
__global__ void smo_kernel_initial(float *d_x, int *d_y, float *d_e, float *d_alpha, int *d_Iup, int *d_Ilow, int numOfData, int numOfAttr,float C ){
   if(initial){
       // value preparation
       float C_local = C;
       int index = blockIdx.x*blockDim.x+threadIdx.x;
       extern __shared__ float x_data[blockDim.x*numOfAttr];
       __shared__ int y_data[blockDim.x];
       __shared__ float e[blockDim.x];
       __shared__ float alpha[blockDim.x];
       __shared__ int Iup[blockDim.x];
       __shared__ int Ilow[blockDim.x];
 
       // put data from global memory to shared memory
       if(index<numOfData){
           for(int i=0;i<numOfAttr;i++){
               x_data[numOfAttr*threadIdx.x+i] = d_x[numOfAttr*index+i];
           }
           y_data[threadIdx.x] = d_y[index];
           e[threadIdx.x] = -y_data[threadIdx.x];
           alpha[threadIdx.x] = 0;
           Iup[threadIdx.x] = divideGroup(y_data[threadIdx.x], alpha[threadIdx.x], C_local,true); // true: detect up; false: detect low
           Ilow[threadIdx.x] = divideGroup(y_data[threadIdx.x], alpha[threadIdx.x], C_local,false);// if it is up/low, the value is 1, otherwise 0;
 
       }
 
   }
 
}
 
__device__ int divideGroup(int y, float alpha, float C, bool isUp){
   if(alpha>0 && alpha<C){
       return 1;
   }
   int res;
   if(isUp){
       if((y==1&&alpha==0)||(y==-1&&alpha==C)){
           res = 1;
       }else{
           res = 0;
       }
       return res;
   }else{
       if((y==1&&alpha==C)||(y==-1&&alpha==0)){
           res = 1;
       }else{
           res = 0;
       }
       return res;
   }
  
 
}
