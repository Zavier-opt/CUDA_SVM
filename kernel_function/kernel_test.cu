#include <stdio.h>
#include <vector>
#include <algorithm>
#include <iostream>
#include "smo_kernel.cu"
 
using namespace std;
 
int main(){
 
   // In host, initialize a 2d matrix h_x (using a vector with size in numOfData*numOfAttr );
   // parameters:
   float C = 0.1;
   // data:
   int numOfData = 10;
   int numOfAttr = 5;
   size_t bytesOfX = numOfData*numOfAttr*sizeof(float);
   size_t bytesOfY = numOfData*sizeof(int);
   vector<float> h_x (numOfData*numOfAttr);
   vector<int> h_y (numOfData);
 
   // initialize h_x, h_y
   generate(h_x.begin(), h_x.end(), [](){return rand()%100;});
   generate(h_y.begin(), h_y.end(), [](){return (rand()%2)*2-1;});
 
 
   // In device, allocate memory to device, transform from host to device
   int *d_x, *d_y;
   hipMalloc(&d_x, bytesOfX);    // hipMalloc(void**, devPtr, size_t, size)
   hipMalloc(&d_y, bytesOfY);
 
   hipMemcpy(d_x, h_x.data(), bytesOfX, hipMemcpyHostToDevice);
   hipMemcpy(d_y, h_y.data(), bytesOfY, hipMemcpyHostToDevice);
 
 
   //
   // test
   // for(int i=0; i<h_y.size();i++){
   //     cout<<h_y[i]<<",";
   // }
    // define num of threads and num of blocks
   int THREADS = 512;
   int BLOCKS;
   if(numOfData%THREADS==0){
       BLOCKS = numOfData/THREADS;
   }else{
       BLOCKS = numOfData/THREADS+1;
   }
 
   bool initial = 1;
   smo_kernel<<BLOCKS, THREADS>>(d_x,d_y,initial,numOfData,numOfAttr);
 
   // free memory
   hipFree(d_x);
   hipFree(d_y);
 
}
