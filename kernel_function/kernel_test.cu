#include <stdio.h>
# include<stdlib.h>
#include <vector>
#include <algorithm>
#include <iostream>
# include <string.h>
#include "smo_kernel.cu"
#include "helper.cu"
#include "lru_list.cu"
#include "lru_list.h"

using namespace std;

    
int main(){
    
    // Read the File
    FILE* fp = fopen("..\\Dataset/MUSK_testing.csv", "r"); // rt means reading only
    if (fp == NULL) {
        printf("Can not open the file!\n");
        return 2;
    }
    else {
        printf("Sucdessfully open the file!\n");
    }

    // Get the size 
    int DATASIZE;
    int FEATURENUM;
    char* line, * record;
    char buffer[4096];
    if ((fp = fopen("..\\Dataset/MUSK_testing.csv", "r")) != NULL) {
        fseek(fp, 0, SEEK_SET);  //��λ���ڶ��У�ÿ��Ӣ���ַ���СΪ1
        char delims[] = ",";
        char* result = NULL;
        int j = 0;
        int i = 0;
        float record_float;
        while ((line = fgets(buffer, sizeof(buffer), fp)) != NULL) {//��û�ж�ȡ���ļ�ĩβʱѭ������{
            record = strtok(line, ",");

            if (i == 0) {
                while (record != NULL)//��ȡÿһ�е�����
                {
                    if (strcmp(record, "Ps:") == 0)//����ȡ��Ps��һ��ʱ�����ټ�����ȡ
                        return 0;
                    //printf("%i, %i, %s \n", i, j, record);//����ȡ����ÿһ�����ݴ�ӡ����
                    record = strtok(NULL, ",");
                    j++;
                    FEATURENUM = j - 1;
                }
            }
            //printf("\n");
            i++;
            j = 0;

        }
        fclose(fp);
        fp = NULL;
        DATASIZE = i - 1;

    }

    // Make the Declaration
    // float  X[DATASIZE][FEATURENUM];
    // float X_flat[DATASIZE*FEATURENUM];
    // int y[DATASIZE];
    vector<float> h_x(DATASIZE * FEATURENUM);
    vector<int> h_y(DATASIZE);

    //char *line,*record;
    //char buffer[4096];
    if ((fp = fopen("..\\Dataset/MUSK_testing.csv", "r")) != NULL) {
        fseek(fp, 0, SEEK_SET);  //��λ���ڶ��У�ÿ��Ӣ���ַ���СΪ1
        char delims[] = ",";
        char* result = NULL;
        int j = 0;
        int i = 0;
        float record_float;
        while ((line = fgets(buffer, sizeof(buffer), fp)) != NULL){//��û�ж�ȡ���ļ�ĩβʱѭ������{
            record = strtok(line, ",");
            while (record != NULL)//��ȡÿһ�е�����
            {
                if (strcmp(record, "Ps:") == 0)//����ȡ��Ps��һ��ʱ�����ټ�����ȡ
                    return 0;
                //printf("%i, %i, %s \n", i, j, record);//����ȡ����ÿһ�����ݴ�ӡ����

                if (i > 0 && j >= 0) {
                    if (j != 166) {
                        record_float = atof(record);
                        // X[i-1][j] = record_float;
                        //printf("%i, %i, %3f \n", i, j, record_float);

                        int index;
                        index = (i - 1) * 166 + j;
                        h_x[index] = record_float;
                    }
                    else {
                        record_float = atof(record);
                        if (record_float == 0.0)
                            record_float = -1;
                        h_y[i - 1] = record_float; // store label
                        //printf("%i, %i, %3f \n", i, j, y[i-1]);
                    }
                }

                //if (j == 167)  //ֻ���ȡǰ167��
                if (j == 166)
                    break;
                record = strtok(NULL, ",");
                j++;
            }
            //printf("\n");
            i++;
            j = 0;

        }
        fclose(fp);
        fp = NULL;

    }

    //cout << DATASIZE << endl;



    // In host, initialize a 2d matrix h_x (using a vector with size in numOfData*numOfAttr );
    // Input parameters:
    float C = 10;
    float slack = 0.1;
    char kernel_function_name[4] = "RBF";
    float Gamma = 0.001;


    // In host:
    int numOfData = DATASIZE;
    int numOfAttr = FEATURENUM;
    size_t bytesOfX = numOfData*numOfAttr*sizeof(float);
    size_t bytesOfY = numOfData*sizeof(int);
    //vector<float> h_x (numOfData*numOfAttr);
    //vector<int> h_y (numOfData);
    vector<float> h_e(numOfData);
    vector<float> h_alpha(numOfData);
    vector<int> h_Iup(numOfData);
    vector<int> h_Ilow(numOfData);
    
    // Initialize LRU List
    head = (struct node *)malloc(sizeof(struct node));
    head->next = NULL;
    
    //generate(h_x.begin(), h_x.end(), [](){return rand()%100;});
    //generate(h_y.begin(), h_y.end(), [](){return (rand()%2)*2-1;});
        
    // In device, allocate memory to device, transform from host to device
    float *d_x;
    int *d_y;
    float *d_e;
    float *d_alpha;
    int *d_Iup;
    int *d_Ilow;
    float *kernel_value;
    int numOfRowInKernel = LRUSIZE;
    size_t bytesOfe = numOfData*sizeof(float);
    size_t bytesOfalpha = numOfData*sizeof(float);
    size_t bytesOfIup = numOfData*sizeof(int);
    size_t bytesOfIlow = numOfData*sizeof(int);
    size_t bytesOfkernel_value = numOfData*numOfRowInKernel*sizeof(float); // kernel value: [k1:[x1,x2,...],k2:[x1,x2,...]]


    hipMalloc(&d_x, bytesOfX);
    hipMalloc(&d_y, bytesOfY);
    hipMalloc(&d_e, bytesOfe);
    hipMalloc(&d_alpha, bytesOfalpha);
    hipMalloc(&d_Iup, bytesOfIup);
    hipMalloc(&d_Ilow, bytesOfIlow);
    hipMalloc(&kernel_value,bytesOfkernel_value); // allocate memory of kernel value in device
    
    // from host to device
    hipMemcpy(d_x, h_x.data(), bytesOfX, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y.data(), bytesOfY, hipMemcpyHostToDevice);
    
    int THREADS = 512;
    int BLOCKS;
    if(numOfData%THREADS==0){
        BLOCKS = (int)numOfData/THREADS;
    }else{
        BLOCKS = (int)numOfData/THREADS+1;
    } 

    cout << numOfData << endl;
    cout << BLOCKS << endl;

    // Go to device, Initial value
    smo_kernel_initial<<<BLOCKS, THREADS>>>(d_x,d_y,d_e, d_alpha,d_Iup, d_Ilow, numOfData,numOfAttr, C);
    hipDeviceSynchronize();



    // Back to host
    hipMemcpy(h_e.data(), d_e, bytesOfe, hipMemcpyDeviceToHost);
    hipMemcpy(h_Ilow.data(),d_Ilow, bytesOfIlow, hipMemcpyDeviceToHost);
    hipMemcpy(h_Iup.data(),d_Iup, bytesOfIup, hipMemcpyDeviceToHost);
    

     // Find low, up vector
    int low=-1;
    int up=-1;
    findSupportVector(h_e, h_Ilow, h_Iup, &low, &up);

    //cout << low << endl;
    //cout << up << endl;

    // First two nodes in LRU list
    struct node* second = (struct node *)malloc(sizeof(struct node));
    head->id = low;
    head->next = second;
    second->id = up;
    second->next = NULL;
    
    int row_low;
    int row_up;

    int iterations = 0;
    // Loop begin
    while(1){
        
        int temp;
        printf("next interations:");
        scanf("%d", &temp);

        //cout << "support vector index:" << endl;
        //cout << up << endl;
        //cout << low << endl;

        float bup = h_e[up];
        float blow = h_e[low];
        float diff = blow - bup;
        //cout << "b value:" << endl;
        //cout<< bup <<endl;
        //cout << blow << endl;

        iterations++;
        if (iterations % 20 == 1) {
            printf("Iteration: %d\n", iterations);
            printf("blow-bup: %0.2f\n", diff);
        }

        if(blow<=bup+2*slack){
            break;
        }
        // Go to device
        // Need LRU function:
        // Input: up, low (they are index of support vector);
        // Output: 
        // cal_low/up (whether we should calculate them)
        // row_low/up (the index of row in kernel values matrix)
        //row_low = 0; // if cal_low/up is true, row_low/up is the posOfRow they should be
        //row_up = 1;// if cal_low/up is false, row_low/up is the posOfRow they have been
        bool cal_low = true;
        bool cal_up = true;
 
        //row_low = push_id(low, head, &cal_low);
        //row_up = push_id(up, head, &cal_up);
        row_low = 0;
        row_up = 1;

        //cout << "support vector row value:" << endl;
        //cout << row_low << endl;
        //cout << row_up << endl;
        //cout << "support vector row status:" << endl;
        //cout << cal_low << endl;
        //cout << cal_up << endl;
      
        //cout << "blocks,threads:" << endl;
        //cout << BLOCKS << endl;
        //cout << THREADS << endl;
        calculate_kernel_update_alpha<<<BLOCKS, THREADS>>>(low, up, kernel_value,d_x, d_y, d_e, d_alpha,d_Iup, d_Ilow, numOfData,numOfAttr, cal_low,cal_up,row_low,row_up,kernel_function_name,Gamma,C);
        hipDeviceSynchronize();
        // 1. get kernel value
        
        // 2. compute alpha, e and Iup Ilow

        // Go Back to host
        hipMemcpy(h_e.data(), d_e, bytesOfe, hipMemcpyDeviceToHost);
        hipMemcpy(h_alpha.data(), d_alpha, bytesOfalpha, hipMemcpyDeviceToHost);
        hipMemcpy(h_Ilow.data(),d_Ilow, bytesOfIlow, hipMemcpyDeviceToHost);
        hipMemcpy(h_Iup.data(),d_Iup, bytesOfIup, hipMemcpyDeviceToHost);
        //cout << "h_alpha values:" << endl;
        //for (int i = 0; i < numOfData; i++) {
        //    cout << h_alpha[i] << " ";
        //}
        //cout << "h_e values:" << endl;
        //for (int i = 0; i < numOfData; i++) {
        //    cout << h_e[i] << " ";
        //}
        //cout << "h_Ilow values:" << endl;
        //for (int i = 0; i < numOfData; i++) {
        //    cout << h_Ilow[i] << " ";
        //}
        //cout << "h_Iup values:" << endl;
        //for (int i = 0; i < numOfData; i++) {
        //    cout << h_Iup[i] << " ";
        //}

        // Find low, up vector
        findSupportVector(h_e, h_Ilow, h_Iup, &low, &up);

    }

    // recieve results
    hipMemcpy(h_alpha.data(), d_alpha,bytesOfalpha,hipMemcpyDeviceToHost);
    vector<float> h_kerel_up(numOfData);
    vector<float> h_kernel_low(numOfData);
    float *d_kernel_up;
    float *d_kernel_low;
    size_t bytesOfSupportKernel = numOfData*sizeof(float);
    hipMalloc(&d_kernel_up,bytesOfSupportKernel);
    hipMalloc(&d_kernel_low,bytesOfSupportKernel);
    write_kernel_to_memory<<<BLOCKS, THREADS>>>(d_kernel_up,d_kernel_low,row_up,row_low,kernel_value,numOfData);
    hipDeviceSynchronize();
    hipMemcpy(h_kerel_up.data(),d_kernel_up,bytesOfSupportKernel,hipMemcpyDeviceToHost);
    hipMemcpy(h_kernel_low.data(),d_kernel_low,bytesOfSupportKernel,hipMemcpyDeviceToHost);
    
    vector<int> predict_res(numOfData);
    make_prediction(predict_res,low,up, numOfData,C, h_kerel_up, h_kernel_low, h_alpha, h_y);

    //cout << "predict:" << endl;
    //for (int i = 0; i < numOfData; i++) {
    //    cout << predict_res[i] << " ";
    //}
    //cout << " " << endl;
    cout << "h_alpha:" << endl;
    for (int i = 0; i < numOfData; i++) {
       cout << h_alpha[i] << " ";
    }
    //cout << " " << endl;

    float training_accuracy = accuracy(predict_res, h_y, numOfData);
    printf("accuracy: %.2f", training_accuracy);
    // Output results for prediction:
    // low, up kernel values:   h_kerel_up, h_kernel_low;
    // low, up support vector:  row_low, row_up;
    // alhps:                   h_alpha;
    // final group:             h_Ilow, h_Iup;

    // Next step: use parameters to make prediction

    // free memory

    // cout<<"h_alpha:"<<endl;
    // for(int i=0;i<numOfData;i++){
    //     cout<<h_alpha[i]<<endl;
    // }

    // cout<<"kernel up:"<<endl;
    // for(int i=0;i<numOfData;i++){
    //     cout<<h_kerel_up[i]<<endl;
    // }
    // cout<<"\n";
    // cout<<"kernel up:"<<endl;
    // for(int i=0;i<numOfData;i++){
    //     cout<<h_kerel_up[i]<<endl;
    // }

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_e);
    hipFree(d_alpha);
    hipFree(d_Iup);
    hipFree(d_Ilow);
    hipFree(kernel_value); 
    
}

