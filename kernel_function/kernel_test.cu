#include <stdio.h>
#include <vector>
#include <algorithm>
#include <iostream>
#include "smo_kernel.cu"
#include "helper.cu"
#include "lru_list.cu"
#include "lru_list.h"
//#include "support.h"

using namespace std;

    
int main(){
    
    // In host, initialize a 2d matrix h_x (using a vector with size in numOfData*numOfAttr );
    // Input parameters:
    float C = 0.1;
    float slack = 0.1;
    char kernel_function_name[4] = "RBF";
    float Gamma = 0.01;

    // In host:
    int numOfData = 10;
    int numOfAttr = 5;
    size_t bytesOfX = numOfData*numOfAttr*sizeof(float);
    size_t bytesOfY = numOfData*sizeof(int);
    vector<float> h_x (numOfData*numOfAttr);
    vector<int> h_y (numOfData);
    vector<float> h_e(numOfData);
    vector<float> h_alpha(numOfData);
    vector<int> h_Iup(numOfData);
    vector<int> h_Ilow(numOfData);
    
    // Initialize LRU List
    head = (struct node *)malloc(sizeof(struct node));
    head->next = NULL;
    
    generate(h_x.begin(), h_x.end(), [](){return rand()%100;});
    generate(h_y.begin(), h_y.end(), [](){return (rand()%2)*2-1;});
        
    // In device, allocate memory to device, transform from host to device
    float *d_x;
    int *d_y;
    float *d_e;
    float *d_alpha;
    int *d_Iup;
    int *d_Ilow;
    float *kernel_value;
    int numOfRowInKernel = LRUSIZE;
    size_t bytesOfe = numOfData*sizeof(float);
    size_t bytesOfalpha = numOfData*sizeof(float);
    size_t bytesOfIup = numOfData*sizeof(int);
    size_t bytesOfIlow = numOfData*sizeof(int);
    size_t bytesOfkernel_value = numOfData*numOfRowInKernel*sizeof(float); // kernel value: [k1:[x1,x2,...],k2:[x1,x2,...]]
    // hipError_t cuda_ret;
    // cuda_ret = hipMalloc(&d_x, bytesOfX);    // hipMalloc(void**, devPtr, size_t, size)
    // if(cuda_ret!=hipSuccess){
    //     cout<<"wrong"<<endl;
    // }
    hipMalloc(&d_x, bytesOfX);
    hipMalloc(&d_y, bytesOfY);
    hipMalloc(&d_e, bytesOfe);
    hipMalloc(&d_alpha, bytesOfalpha);
    hipMalloc(&d_Iup, bytesOfIup);
    hipMalloc(&d_Ilow, bytesOfIlow);
    hipMalloc(&kernel_value,bytesOfkernel_value); // allocate memory of kernel value in device
    
    // from host to device
    hipMemcpy(d_x, h_x.data(), bytesOfX, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y.data(), bytesOfY, hipMemcpyHostToDevice);
    
    int THREADS = 512;
    int BLOCKS;
    if(numOfData%THREADS==0){
        BLOCKS = (int)numOfData/THREADS;
    }else{
        BLOCKS = (int)numOfData/THREADS+1;
    } 

    hipDeviceSynchronize();
    // Go to device, Initial value
    smo_kernel_initial<<<BLOCKS, THREADS>>>(d_x,d_y,d_e, d_alpha,d_Iup, d_Ilow, numOfData,numOfAttr, C);
    hipDeviceSynchronize();



    // Back to host
    hipMemcpy(h_e.data(), d_e, bytesOfe, hipMemcpyDeviceToHost);
    hipMemcpy(h_Ilow.data(),d_Ilow, bytesOfIlow, hipMemcpyDeviceToHost);
    hipMemcpy(h_Iup.data(),d_Iup, bytesOfIup, hipMemcpyDeviceToHost);
    
    for(int i=0;i<numOfData;i++){
        cout<<h_e[i]<<endl;
    }  

     // Find low, up vector
    int low=-1;
    int up=-1;
    findSupportVector(h_e, h_Ilow, h_Iup, &low, &up);



    // First two nodes in LRU list
    struct node* second = (struct node *)malloc(sizeof(struct node));
    head->id = low;
    head->next = second;
    second->id = up;
    second->next = NULL;
    
    int row_low;
    int row_up;

    // Loop begin
    while(1){
        //cout<<1<<endl;
        float bup = h_e[up];
        float blow = h_e[low];
        if(blow<=bup+2*slack){
            break;
        }
        // Go to device
        // Need LRU function:
        // Input: up, low (they are index of support vector);
        // Output: 
        // cal_low/up (whether we should calculate them)
        // row_low/up (the index of row in kernel values matrix)
        //row_low = 0; // if cal_low/up is true, row_low/up is the posOfRow they should be
        //row_up = 1;// if cal_low/up is false, row_low/up is the posOfRow they have been
        bool cal_low = true;
        bool cal_up = true;
        row_low = push_id(low, head, &cal_low);
        row_up = push_id(up, head, &cal_up);
      
        calculate_kernel_update_alpha<<<BLOCKS, THREADS>>>(low, up, kernel_value,d_x, d_y, d_e, d_alpha,d_Iup, d_Ilow, numOfData,numOfAttr, cal_low,cal_up,row_low,row_up,kernel_function_name,Gamma,C);
        // 1. get kernel value
        
        // 2. compute alpha, e and Iup Ilow

        // Go Back to host
        hipMemcpy(h_e.data(), d_e, bytesOfe, hipMemcpyDeviceToHost);
        hipMemcpy(h_Ilow.data(),d_Ilow, bytesOfIlow, hipMemcpyDeviceToHost);
        hipMemcpy(h_Iup.data(),d_Iup, bytesOfIup, hipMemcpyDeviceToHost);

        // Find low, up vector
        findSupportVector(h_e, h_Ilow, h_Iup, &low, &up);

    }

    // recieve results
    hipMemcpy(h_alpha.data(), d_alpha,bytesOfalpha,hipMemcpyDeviceToHost);
    vector<float> h_kerel_up(numOfData);
    vector<float> h_kernel_low(numOfData);
    float *d_kernel_up;
    float *d_kernel_low;
    size_t bytesOfSupportKernel = numOfData*sizeof(float);
    hipMalloc(&d_kernel_up,bytesOfSupportKernel);
    hipMalloc(&d_kernel_low,bytesOfSupportKernel);
    write_kernel_to_memory<<<BLOCKS, THREADS>>>(d_kernel_up,d_kernel_low,row_up,row_low,kernel_value,numOfData);
    hipMemcpy(h_kerel_up.data(),d_kernel_up,bytesOfSupportKernel,hipMemcpyDeviceToHost);
    hipMemcpy(h_kernel_low.data(),d_kernel_low,bytesOfSupportKernel,hipMemcpyDeviceToHost);
 
    // Output results for prediction:
    // low, up kernel values:   h_kerel_up, h_kernel_low;
    // low, up support vector:  row_low, row_up;
    // alhps:                   h_alpha;
    // final group:             h_Ilow, h_Iup;

    // Next step: use parameters to make prediction

    // free memory

    // cout<<"h_alpha:"<<endl;
    // for(int i=0;i<numOfData;i++){
    //     cout<<h_alpha[i]<<endl;
    // }

    // cout<<"kernel up:"<<endl;
    // for(int i=0;i<numOfData;i++){
    //     cout<<h_kerel_up[i]<<endl;
    // }
    // cout<<"\n";
    // cout<<"kernel up:"<<endl;
    // for(int i=0;i<numOfData;i++){
    //     cout<<h_kerel_up[i]<<endl;
    // }

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_e);
    hipFree(d_alpha);
    hipFree(d_Iup);
    hipFree(d_Ilow);
    hipFree(kernel_value); 
    
}

