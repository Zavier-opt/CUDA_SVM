#include <stdio.h>
#include <vector>
#include <algorithm>
#include <iostream>
#include "smo_kernel.cu"
#include "helper.cu"

using namespace std;
    
int main(){
    
    // In host, initialize a 2d matrix h_x (using a vector with size in numOfData*numOfAttr );
    // parameters:
    float C = 0.1;
    float slack = 0.1;

    // In host:
    int numOfData = 10;
    int numOfAttr = 5;
    size_t bytesOfX = numOfData*numOfAttr*sizeof(float);
    size_t bytesOfY = numOfData*sizeof(int);
    vector<float> h_x (numOfData*numOfAttr);
    vector<int> h_y (numOfData);
    vector<float> h_e(numOfData);
    vector<float> h_Iup(numOfData);
    vector<float> h_Ilow(numOfData);
    
    
    // initialize h_x, h_y
    generate(h_x.begin(), h_x.end(), [](){return rand()%100;});
    generate(h_y.begin(), h_y.end(), [](){return (rand()%2)*2-1;});
    
    
    // In device, allocate memory to device, transform from host to device
    float *d_x;
    int *d_y;
    float *d_e;
    float *d_alpha;
    int *d_Iup;
    int *d_Ilow;
    float *kernel_value;
    int numOfRowInKernel = 500;
    size_t bytesOfe = numOfData*sizeof(float);
    size_t bytesOfalpha = numOfData*size(float);
    size_t bytesOfIup = numOfData*sizeof(int);
    size_t bytesOfIlow = numOfData*sizeof(int);
    size_t bytesOfkernel_value = numOfData*numOfRowInKernel*sizeof(float);
    hipMalloc(&d_x, bytesOfX);    // hipMalloc(void**, devPtr, size_t, size)
    hipMalloc(&d_y, bytesOfY);
    hipMalloc(&d_e, bytesOfe);
    hipMalloc(&d_alpha, bytesOfalpha);
    hipMalloc(&d_Iup, bytesOfIup);
    hipMalloc(&d_Ilow, bytesOfIlow);
    hipMalloc(&kernel_value,bytesOfkernel_value); // allocate memory of kernel value in device
    
    // from host to device
    hipMemcpy(d_x, h_x.data(), bytesOfX, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y.data(), bytesOfY, hipMemcpyHostToDevice);
    
    //
    // test
    // for(int i=0; i<h_y.size();i++){
    //     cout<<h_y[i]<<",";
    // }
        // define num of threads and num of blocks
    int THREADS = 512;
    int BLOCKS;
    if(numOfData%THREADS==0){
        BLOCKS = numOfData/THREADS;
    }else{
        BLOCKS = numOfData/THREADS+1;
    }

    // Go to device, Initial value
    smo_kernel_initial<<BLOCKS, THREADS>>(d_x,d_y,d_e, d_alpha,d_Iup, d_Ilow, numOfData,numOfAttr, C);
    
    // Back to host
    hipMemcpy(h_e.data(), d_e, bytesOfe, hipMemcpyDeviceToHost);
    hipMemcpy(h_Ilow.data(),d_Ilow, bytesOfIlow, hipMemcpyDeviceToHost);
    hipMemcpy(h_Iup.data(),d_Iup, bytesOfIup, hipMemcpyDeviceToHost);
    
     // Find low, up vector
    int low=-1;
    int up=-1;
    findSupportVector(h_e, h_Ilow, h_Iup, &low, &up);

    // Loop begin
    while(1){
        float bup = h_e[up];
        float blow = h_e[low];
        if(blow<=bup+2*slack){
            break;
        }
        // Go to device

        // 1. get kernel value

        // 2. compute alpha, e and Iup Ilow

        // Back to host
        

    }

    

    // free memory
    hipFree(d_x);
    hipFree(d_y);
    
}

