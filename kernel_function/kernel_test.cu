#include <stdio.h>
#include <vector>
#include <algorithm>
#include <iostream>
#include "smo_kernel.cu"
#include "helper.cu"

using namespace std;
    
int main(){
    
    // In host, initialize a 2d matrix h_x (using a vector with size in numOfData*numOfAttr );
    // parameters:
    float C = 0.1;
    // In host:
    int numOfData = 10;
    int numOfAttr = 5;
    size_t bytesOfX = numOfData*numOfAttr*sizeof(float);
    size_t bytesOfY = numOfData*sizeof(int);
    vector<float> h_x (numOfData*numOfAttr);
    vector<int> h_y (numOfData);
    vector<float> h_e(numOfData);
    vector<float> h_Iup(numOfData);
    vector<float> h_Ilow(numOfData);
    
    
    // initialize h_x, h_y
    generate(h_x.begin(), h_x.end(), [](){return rand()%100;});
    generate(h_y.begin(), h_y.end(), [](){return (rand()%2)*2-1;});
    
    
    // In device, allocate memory to device, transform from host to device
    float *d_x;
    int *d_y;
    float *d_e;
    float *d_alpha;
    int *d_Iup;
    int *d_Ilow;
    size_t bytesOfe = numOfData*sizeof(float);
    size_t bytesOfalpha = numOfData*size(float);
    size_t bytesOfIup = numOfData*sizeof(int);
    size_t bytesOfIlow = numOfData*sizeof(int);
    hipMalloc(&d_x, bytesOfX);    // hipMalloc(void**, devPtr, size_t, size)
    hipMalloc(&d_y, bytesOfY);
    hipMalloc(&d_e, bytesOfe);
    hipMalloc(&d_alpha, bytesOfalpha);
    hipMalloc(&d_Iup, bytesOfIup);
    hipMalloc(&d_Ilow, bytesOfIlow);
    
        // from host to device
    hipMemcpy(d_x, h_x.data(), bytesOfX, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y.data(), bytesOfY, hipMemcpyHostToDevice);
    
    //
    // test
    // for(int i=0; i<h_y.size();i++){
    //     cout<<h_y[i]<<",";
    // }
        // define num of threads and num of blocks
    int THREADS = 512;
    int BLOCKS;
    if(numOfData%THREADS==0){
        BLOCKS = numOfData/THREADS;
    }else{
        BLOCKS = numOfData/THREADS+1;
    }

    // Initial value
    smo_kernel_initial<<BLOCKS, THREADS>>(d_x,d_y,d_e, d_alpha,d_Iup, d_Ilow, numOfData,numOfAttr, C);
    
    // from device to host
    hipMemcpy(h_e.data(), d_e, bytesOfe, hipMemcpyDeviceToHost);
    hipMemcpy(h_Ilow.data(),d_Ilow, bytesOfIlow, hipMemcpyDeviceToHost);
    hipMemcpy(h_Iup.data(),d_Iup, bytesOfIup, hipMemcpyDeviceToHost);
    
     // Find low, up vector
    int low=-1;
    int up=-1;
    findSupportVector(h_e, h_Ilow, h_Iup, &low, &up);

    // free memory
    hipFree(d_x);
    hipFree(d_y);
    
}

