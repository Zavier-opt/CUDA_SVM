
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

using namespace std;

__global__ void myKernel() 
{ 
  printf("Hello, world from the device!\n"); 
} 

int main() 
{ 
  myKernel<<<1,10>>>(); 
  hipDeviceSynchronize();
  cout<<1<<endl;
} 